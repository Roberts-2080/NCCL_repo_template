#include "common.h"


void profileCopies(float        *h_a, 
                   float        *h_b, 
                   float        *d, 
                   unsigned int n,
                   const char   *desc) {

	  printf("\n%s transfers\n", desc);

	  unsigned int bytes = n * sizeof(float);

	  // events for timing
	  hipEvent_t startEvent, stopEvent; 

	  checkCuda( hipEventCreate(&startEvent) );
	  checkCuda( hipEventCreate(&stopEvent) );

	  checkCuda( hipEventRecord(startEvent, 0) );
	  checkCuda( hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice) );
	  checkCuda( hipEventRecord(stopEvent, 0) );
	  checkCuda( hipEventSynchronize(stopEvent) );

	  float time;
	  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	  printf("\t\t\tHost to Device >> [Bandwidth (GB/s): %f]\n", bytes * 1e-6 / time);

	  checkCuda( hipEventRecord(startEvent, 0) );
	  checkCuda( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
	  checkCuda( hipEventRecord(stopEvent, 0) );
	  checkCuda( hipEventSynchronize(stopEvent) );

	  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	  printf("\t\t\tDevice to Host >> [Bandwidth (GB/s): %f]\n", bytes * 1e-6 / time);

	  for (int i = 0; i < n; ++i) {
	    if (h_a[i] != h_b[i]) {
	      printf("*** %s transfers failed ***\n", desc);
	      break;
	    }
	  }

	  checkCuda( hipEventDestroy(startEvent) );
	  checkCuda( hipEventDestroy(stopEvent) );
}

int main()
{
	size_t nElements = 1024*1024*32;
	size_t bytes = nElements * sizeof(float);

	// host arrays
	float *h_aPageable, *h_bPageable;   
	float *h_aPinned, *h_bPinned;

	float *d_a;

	h_aPageable = (float*)malloc(bytes);                    // host pageable
	h_bPageable = (float*)malloc(bytes);                    // host pageable
	checkCuda( hipHostMalloc((void**)&h_aPinned, bytes) ); // host pinned
	checkCuda( hipHostMalloc((void**)&h_bPinned, bytes) ); // host pinned
	checkCuda( hipMalloc((void**)&d_a, bytes) );           // device

	for (int i = 0; i < nElements; ++i) h_aPageable[i] = i;      
	memcpy(h_aPinned, h_aPageable, bytes);
	memset(h_bPageable, 0, bytes);
	memset(h_bPinned, 0, bytes);


	printf("\n=============PINNED/PAGED TRANSFER BENCHMARK=================\n\n");

	// perform copies and report bandwidth

	for (size_t i = 4; i <= nElements; i*=2) {
		printf("\tPinned/Paged transfer on %ld bytes",i*sizeof(float));
		profileCopies(h_aPageable, h_bPageable, d_a, i, "\t\tPageable");
		profileCopies(h_aPinned, h_bPinned, d_a, i, "\t\tPinned");
	}


	hipFree(d_a);
	hipHostFree(h_aPinned);
	hipHostFree(h_bPinned);
	free(h_aPageable);
	free(h_bPageable);

	return 0;
};
