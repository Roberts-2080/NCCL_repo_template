#include "common.h" 

void run(float* h_a, float* d_a, size_t bytes) {
 	float _time_;
	hipEvent_t startEvent, stopEvent; 
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));

	checkCuda(hipEventRecord(startEvent, 0));
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	checkCuda( hipEventRecord(stopEvent, 0));
	checkCuda( hipEventSynchronize(stopEvent));
	checkCuda( hipEventElapsedTime(&_time_, startEvent, stopEvent)); 
	printf("\t\tHost to Device >> ");
	printf("[Bandwidth (GB/s): %f]\n", bytes * 1e-6 / _time_);

	checkCuda(hipEventRecord(startEvent, 0));
	hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
	checkCuda( hipEventRecord(stopEvent, 0));
	checkCuda( hipEventSynchronize(stopEvent));
	checkCuda( hipEventElapsedTime(&_time_, startEvent, stopEvent)); 
	printf("\t\tDevice to Host >> ");
	printf("[Bandwidth (GB/s): %f]\n", bytes * 1e-6 / _time_);

	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
}

int main()
{
	size_t N = 1024*1024*32;
	size_t bytes = N * sizeof(float);
	float *h_a = (float*)malloc(bytes);
	float *d_a;
	hipMalloc((float**)&d_a, bytes);
	printf("\n====================CUDA MEMCPY BENCHMARK====================\n\n");

	memset(h_a, 420.69, bytes);

	for (size_t i = 4; i <= bytes; i*=2) {
		printf("\tcudaMemcpy on %ld bytes\n", i);
		run(h_a, d_a, i);
	}

	return 0;
}
