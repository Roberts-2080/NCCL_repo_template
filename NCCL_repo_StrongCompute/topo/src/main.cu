#include "common.h"

// Taken from: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html

int main(int argc, char* argv[]) {

	ncclComm_t comms[2];

	//managing 2 device
	int nDev = 2;
	int size = 32 * 1024 * 1024;
	int devs[2] = {0,1};

	//allocating and initializing device buffers
	float** sendbuff = (float**)malloc(nDev * sizeof(float*));
	float** recvbuff = (float**)malloc(nDev * sizeof(float*));
	hipStream_t* s = (hipStream_t *) malloc(sizeof(hipStream_t) * nDev);

	for (int i = 0; i < nDev; ++i) {
		CUDACHECK(hipSetDevice(i));
		CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
		CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
		CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
		CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
		CUDACHECK(hipStreamCreate(s+i));
	}

	printf("Initialize NCCL\n");
	//initializing NCCL
	NCCLCHECK(ncclCommInitAll(comms, nDev, devs));


	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	checkCuda(hipEventRecord(startEvent, 0));


	// calling NCCL communication API. Group API is required hwne using
	// multiple devices per thread
	printf("Calling NCCL All Reduce\n");
	NCCLCHECK(ncclGroupStart());
	for (int i = 0; i < nDev; ++i) {
		NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum, comms[i], s[i]));
	}
	NCCLCHECK(ncclGroupEnd());



	checkCuda( hipEventRecord(stopEvent, 0));
	checkCuda( hipEventSynchronize(stopEvent));

	//synchronizing on CUDA streams to wait for completion of NCCL operation
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	printf("Synchronize\n");
	for (int i = 0; i < nDev; ++i) {
		printf("\tset device %d\n", i);
		CUDACHECK(hipSetDevice(i));
		printf("\tstream synchronize %d\n", i);
		CUDACHECK(hipStreamSynchronize(s[i]));
	}

	printf("Free\n");
	//free device buffers
	for (int i = 0; i < nDev; ++i) {
		CUDACHECK(hipSetDevice(i));
		CUDACHECK(hipFree(sendbuff[i]));
		CUDACHECK(hipFree(recvbuff[i]));
	}


	//finalizing NCCL
	printf("destroying nccl communicators\n");
	for(int i = 0; i < nDev; ++i)
		ncclCommDestroy(comms[i]);

	printf("Success!\n");

	return 0;
}
